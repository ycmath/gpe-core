#include "hip/hip_runtime.h"
## gpe_core/gpu/assemble_graph.cu
## -----------------------------------------------------------
## GPU-side 객체 그래프 조립 커널
##
## * op[]      : 0=NEW, 1=APPEND, 2=REPEAT_BEG, 3=REPEAT_END
## * ida[],idb : 1st / 2nd ID operand (remap 결과)
## * meta_cls  : NEW 행의 class-id (0 dict, 1 list, >=2 custom LUT)
## * meta_key  : APPEND 행의 key-id (lut_key), 0xFFFFFFFF = none
##
## 출력
## * d_type : 0 dict · 1 list · 2 custom
## * d_head : head pointer to first child (single-linked list)
## * d_next : sibling pointer (next child)
## * d_key  : dict 전용 key-id
##/ -----------------------------------------------------------

#include <hip/hip_runtime.h>

extern "C" __global__
void assemble_graph(const uint8_t*  __restrict__ op,
                    const uint32_t* __restrict__ ida,
                    const uint32_t* __restrict__ idb,
                    const uint16_t* __restrict__ meta_cls,
                    const uint32_t* __restrict__ meta_key,
                    uint8_t*  __restrict__ d_type,
                    uint32_t* __restrict__ d_head,
                    uint32_t* __restrict__ d_next,
                    uint32_t* __restrict__ d_key,
                    int n_rows)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n_rows) return;

    uint8_t code = op[i];

    if (code == 0u) {                        // NEW
        uint32_t vid = ida[i];
        uint16_t cls = meta_cls[i];

        // type encoding
        d_type[vid] = (cls == 0u) ? 0u : (cls == 1u ? 1u : 2u);
        d_head[vid] = 0xFFFFFFFFu;           // sentinel null
        d_next[vid] = 0xFFFFFFFFu;
    }
    else if (code == 1u) {                   // APPEND
        uint32_t parent = ida[i];
        uint32_t child  = idb[i];

        // atomic LIFO push: child -> head[parent]
        uint32_t prev = atomicExch(&d_head[parent], child);
        d_next[child] = prev;

        // dict key
        uint32_t k = meta_key[i];
        if (k != 0xFFFFFFFFu)
            d_key[child] = k;
    }
    // REPEAT tokens are structural only – ignored here
}


## * **동기화 필요 없음**: `atomicExch` 로 child 단일-링크 list 구성 → post-pass 에서 역순 iterate.
## * **공유 메모리**: row 단위 prefix 연산이 없으므로 쓰지 않음 → Warp divergence 無.
