#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>

extern "C" __global__
void assemble_graph(const uint8_t*  __restrict__ op,
                    const uint32_t* __restrict__ ida,
                    const uint32_t* __restrict__ idb,
                    const uint16_t* __restrict__ meta_cls,
                    const uint32_t* __restrict__ meta_key,
                    uint8_t*  __restrict__ d_type,
                    uint32_t* __restrict__ d_head,
                    uint32_t* __restrict__ d_next,
                    uint32_t* __restrict__ d_key,
                    int n_rows)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n_rows) return;

    uint8_t code = op[i];

    if (code == 0u) {                        // NEW
        uint32_t vid = ida[i];
        uint16_t cls = meta_cls[i];

        // type encoding
        d_type[vid] = (cls == 0u) ? 0u : (cls == 1u ? 1u : 2u);
        d_head[vid] = 0xFFFFFFFFu;           // sentinel null
        d_next[vid] = 0xFFFFFFFFu;
    }
    else if (code == 1u) {                   // APPEND
        uint32_t parent = ida[i];
        uint32_t child  = idb[i];

        // atomic LIFO push: child -> head[parent]
        uint32_t prev = atomicExch(&d_head[parent], child);
        d_next[child] = prev;

        // dict key
        uint32_t k = meta_key[i];
        if (k != 0xFFFFFFFFu)
            d_key[child] = k;
    }
    // REPEAT tokens are structural only – ignored here
}


## * **동기화 필요 없음**: `atomicExch` 로 child 단일-링크 list 구성 → post-pass 에서 역순 iterate.
## * **공유 메모리**: row 단위 prefix 연산이 없으므로 쓰지 않음 → Warp divergence 無.
