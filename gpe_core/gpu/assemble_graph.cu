
#include <hip/hip_runtime.h>
#include <stdint.h>   // ← 8/16/32-bit 정수 타입 정의

extern "C" __global__
void assemble_graph(const uint8_t*  __restrict__ op,
                    const uint32_t* __restrict__ ida,
                    const uint32_t* __restrict__ idb,
                    const uint16_t* __restrict__ meta_cls,
                    const uint32_t* __restrict__ meta_key,
                    uint8_t*  __restrict__ d_type,
                    uint32_t* __restrict__ d_head,
                    uint32_t* __restrict__ d_next,
                    uint32_t* __restrict__ d_key,
                    int n_rows)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n_rows) return;

    uint8_t code = op[i];

    if (code == 0u) {                        // NEW
        uint32_t vid = ida[i];
        uint16_t cls = meta_cls[i];

        // type encoding
        d_type[vid] = (cls == 0u) ? 0u : (cls == 1u ? 1u : 2u);
        d_head[vid] = 0xFFFFFFFFu;           // sentinel null
        d_next[vid] = 0xFFFFFFFFu;
    }
    else if (code == 1u) {                   // APPEND
        uint32_t parent = ida[i];
        uint32_t child  = idb[i];

        // atomic LIFO push: child -> head[parent]
        uint32_t prev = atomicExch(&d_head[parent], child);
        d_next[child] = prev;

        // dict key
        uint32_t k = meta_key[i];
        if (k != 0xFFFFFFFFu)
            d_key[child] = k;
    }
    // REPEAT tokens are structural only – ignored here
}

